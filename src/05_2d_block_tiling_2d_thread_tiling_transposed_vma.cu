#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "cuda_gemm.hpp"
#include "cuda_utils.hpp"
#include "gemm_utils.cuh"

// kernel
template <
    typename T,
    const size_t BLOCK_TILE_SIZE_M,
    const size_t BLOCK_TILE_SIZE_N,
    const size_t BLOCK_TILE_SIZE_K,
    const size_t THREAD_TILE_SIZE_M,
    const size_t THREAD_TILE_SIZE_N
>
__global__ void gemm_v05_vectorized(size_t m, size_t n, size_t k,
                            const T alpha,
                            const T *A, size_t lda,
                            const T *B, size_t ldb,
                            const T beta,
                            T *C, size_t ldc)
{
    constexpr size_t NUM_THREADS{(BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_N) / (THREAD_TILE_SIZE_M * THREAD_TILE_SIZE_N)};
    constexpr size_t NUM_THREADS_PER_BLOCK_N{BLOCK_TILE_SIZE_N / THREAD_TILE_SIZE_N};

    const size_t thread_linear_idx{threadIdx.y * blockDim.x + threadIdx.x};

    __shared__ T A_thread_block_tile_shared_transposed[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_M];
    __shared__ T B_thread_block_tile_shared[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_N];

    size_t num_AB_thread_block_tiles{(k + BLOCK_TILE_SIZE_K - 1U) / BLOCK_TILE_SIZE_K};

    constexpr size_t NUM_VECTOR_UNITS{sizeof(int4) / sizeof(T)};
    // check if an integer number of units can fit into a vector
    static_assert(sizeof(int4) % sizeof(T) == 0U);

    // check if an integer number of vector can fit into A_thread_block_tile_shared and B_thread_block_tile_shared
    static_assert(BLOCK_TILE_SIZE_K % NUM_VECTOR_UNITS == 0U);
    static_assert(BLOCK_TILE_SIZE_N % NUM_VECTOR_UNITS == 0U);

    // check if an integer number of vector can fit into A_tmp and B_tmp
    constexpr size_t VECTORIZED_THREAD_TILE_SIZE_N{THREAD_TILE_SIZE_N / NUM_VECTOR_UNITS};
    constexpr size_t VECTORIZED_THREAD_TILE_SIZE_M{THREAD_TILE_SIZE_M / NUM_VECTOR_UNITS};
    static_assert(THREAD_TILE_SIZE_N % NUM_VECTOR_UNITS == 0U);
    static_assert(THREAD_TILE_SIZE_M % NUM_VECTOR_UNITS == 0U);

    T sum[THREAD_TILE_SIZE_M][THREAD_TILE_SIZE_N] = {static_cast<T>(0.0)};
    T A_tmp[THREAD_TILE_SIZE_M] = {static_cast<T>(0.0)};
    T B_tmp[THREAD_TILE_SIZE_N] = {static_cast<T>(0.0)};
    
    for (size_t i{0U}; i < num_AB_thread_block_tiles; ++i)
    {
        size_t AB_thread_block_tile_idx{i};

        // copy A, B to shared memory
        load_data_to_shared_memory_transposed_vectorized<T, BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K, NUM_THREADS>(
            A, lda,
            B, ldb,
            A_thread_block_tile_shared_transposed,
            B_thread_block_tile_shared,
            AB_thread_block_tile_idx,
            thread_linear_idx,
            m, n, k
        );

        __syncthreads();

        // compute
        #pragma unroll
        for (size_t j{0U}; j < BLOCK_TILE_SIZE_K; ++j)
        {
            // load necessary A data
            size_t A_thread_block_tile_row_idx{(thread_linear_idx / NUM_THREADS_PER_BLOCK_N) * THREAD_TILE_SIZE_M};
            size_t A_thread_block_tile_col_idx{j};

            #pragma unroll
            for (size_t thread_tile_row_idx{0U}; thread_tile_row_idx < VECTORIZED_THREAD_TILE_SIZE_M; ++thread_tile_row_idx)
            {
                *reinterpret_cast<int4*>(&A_tmp[thread_tile_row_idx*NUM_VECTOR_UNITS])
                    = *reinterpret_cast<const int4*>(
                        &A_thread_block_tile_shared_transposed[A_thread_block_tile_col_idx][A_thread_block_tile_row_idx + thread_tile_row_idx*NUM_VECTOR_UNITS]
                    );
            }

            // load necessary B data
            size_t B_thread_block_tile_row_idx{j};
            size_t B_thread_block_tile_col_idx{(thread_linear_idx % NUM_THREADS_PER_BLOCK_N) * THREAD_TILE_SIZE_N};

            #pragma unroll
            for (size_t thread_tile_col_idx{0U}; thread_tile_col_idx < VECTORIZED_THREAD_TILE_SIZE_N; ++thread_tile_col_idx)
            {
                *reinterpret_cast<int4*>(&B_tmp[thread_tile_col_idx*NUM_VECTOR_UNITS])
                    = *reinterpret_cast<const int4*>(
                        &B_thread_block_tile_shared[B_thread_block_tile_row_idx][B_thread_block_tile_col_idx + thread_tile_col_idx*NUM_VECTOR_UNITS]
                    );
            }

            #pragma unroll
            for (size_t thread_tile_row_idx{0U}; thread_tile_row_idx < THREAD_TILE_SIZE_M; ++thread_tile_row_idx)
            {
                #pragma unroll
                for (size_t thread_tile_col_idx{0U}; thread_tile_col_idx < THREAD_TILE_SIZE_N; ++thread_tile_col_idx)
                {
                    sum[thread_tile_row_idx][thread_tile_col_idx] += A_tmp[thread_tile_row_idx] * B_tmp[thread_tile_col_idx];
                }
            }
        }
        
        __syncthreads();
    }

    // write C to global memory
    #pragma unroll
    for (size_t thread_tile_row_idx{0U}; thread_tile_row_idx < THREAD_TILE_SIZE_M; ++thread_tile_row_idx)
    {   
        #pragma unroll
        for (size_t thread_tile_col_idx{0U}; thread_tile_col_idx < THREAD_TILE_SIZE_N; ++thread_tile_col_idx)
        {
            size_t C_row_idx{(blockIdx.y * BLOCK_TILE_SIZE_M) + thread_linear_idx / NUM_THREADS_PER_BLOCK_N * THREAD_TILE_SIZE_M + thread_tile_row_idx};
            size_t C_col_idx{(blockIdx.x * BLOCK_TILE_SIZE_N) + thread_linear_idx % NUM_THREADS_PER_BLOCK_N * THREAD_TILE_SIZE_N + thread_tile_col_idx};
            if (C_row_idx < m && C_col_idx < n)
            {
                C[C_row_idx * ldc + C_col_idx] = alpha * sum[thread_tile_row_idx][thread_tile_col_idx] + beta * C[C_row_idx * ldc + C_col_idx];
            }
        }
    }
}                            


// launch
template <typename T>
void launch_gemm_kernel_v05_vectorized(size_t m, size_t n, size_t k,
                            const T *alpha,
                            const T *A, size_t lda,
                            const T *B, size_t ldb,
                            const T *beta,
                            T *C, size_t ldc,
                            hipStream_t stream)
{   
    constexpr size_t BLOCK_TILE_SIZE_M{128U};
    constexpr size_t BLOCK_TILE_SIZE_N{128U};
    constexpr size_t BLOCK_TILE_SIZE_K{8U};
    
    constexpr size_t THREAD_TILE_SIZE_M{8U};
    constexpr size_t THREAD_TILE_SIZE_N{8U};

    constexpr size_t NUM_THREADS_PER_BLOCK{(BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_N) / (THREAD_TILE_SIZE_M * THREAD_TILE_SIZE_N)};

    // check if threads can fit into a block (because each thread now is a small column)
    static_assert(BLOCK_TILE_SIZE_M % THREAD_TILE_SIZE_M == 0U);
    static_assert(BLOCK_TILE_SIZE_N % THREAD_TILE_SIZE_N == 0U);

    // check if each thread within a block can be assigned the same workload
    static_assert(BLOCK_TILE_SIZE_M * BLOCK_TILE_SIZE_K % NUM_THREADS_PER_BLOCK == 0U);
    static_assert(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_N % NUM_THREADS_PER_BLOCK == 0U);

    dim3 block{NUM_THREADS_PER_BLOCK, 1U, 1U};
    dim3 grid{
        (static_cast<unsigned int>(n) + static_cast<unsigned int>(BLOCK_TILE_SIZE_N) - 1U) / static_cast<unsigned int>(BLOCK_TILE_SIZE_N),
        (static_cast<unsigned int>(m) + static_cast<unsigned int>(BLOCK_TILE_SIZE_M) - 1U) / static_cast<unsigned int>(BLOCK_TILE_SIZE_M),
        1U
    };

    gemm_v05_vectorized<T, BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K, THREAD_TILE_SIZE_M, THREAD_TILE_SIZE_N>
                <<<grid, block, 0, stream>>>(m, n, k, *alpha, A, lda, B, ldb, *beta, C, ldc);
    
    CHECK_LAST_CUDA_ERROR();
}                            


// explicit instantiation

template void launch_gemm_kernel_v05_vectorized<float>(size_t m, size_t n, size_t k,
                                   const float *alpha,
                                   const float *A, size_t lda,
                                   const float *B, size_t ldb,
                                   const float *beta,
                                   float *C, size_t ldc,
                                   hipStream_t stream);

template void launch_gemm_kernel_v05_vectorized<double>(size_t m, size_t n, size_t k,
                                    const double *alpha,
                                    const double *A, size_t lda,
                                    const double *B, size_t ldb,
                                    const double *beta,
                                    double *C, size_t ldc,
                                    hipStream_t stream);                                   

template void launch_gemm_kernel_v05_vectorized<__half>(size_t m, size_t n, size_t k,
                                   const __half *alpha,
                                   const __half *A, size_t lda,
                                   const __half *B, size_t ldb,
                                   const __half *beta,
                                   __half *C, size_t ldc,
                                   hipStream_t stream);                                   