#include <iostream>
#include <hip/hip_runtime.h>

#include "profile_utils.cuh"
#include "cuda_gemm.hpp"

int main() {

    // print device information
    print_device_info();

    constexpr size_t num_repeats{1U};
    constexpr size_t num_warmups{1U};

    float const fp32_abs_tol{1.0e-3f};
    double const fp32_rel_tol{0.0e-4f};

    constexpr size_t m{4096U};
    constexpr size_t n{4096U};
    constexpr size_t k{4096U};

    constexpr size_t lda{(m + 16U - 1U) / 16U * 16U};
    constexpr size_t ldb{(k + 16U - 1U) / 16U * 16U};
    constexpr size_t ldc{(m + 16U - 1U) / 16U * 16U};

    static_assert(lda >= k);
    static_assert(ldb >= n);
    static_assert(ldc >= n);

    std::cout << "Matrix size: " << m << " x " << n << " x " << k << std::endl;
    std::cout << "Matrix A: " << m << " x " << k << " Leading Dimension Size " << lda << std::endl;
    std::cout << "Matrix B: " << k << " x " << n << " Leading Dimension Size " << ldb << std::endl;
    std::cout << "Matrix C: " << m << " x " << n << " Leading Dimension Size " << ldc << std::endl;

    // Define all the gemm kernel launch functions
    std::vector<
        std::pair<
            std::string, 
            std::function<void(size_t, size_t, size_t, 
                                const float*, 
                                const float*, size_t,
                                const float*, size_t, 
                                const float*,
                                float*, size_t, 
                                hipStream_t)>>> const gemm_kernel_launch_functions {
                                    {"Custom gemm kernel V00", launch_gemm_kernel_v00<float>},
                                    {"Custom gemm kernel V01", launch_gemm_kernel_v01<float>},
                                };

    for (auto gemm_kernel_launch_function : gemm_kernel_launch_functions) {
        std::cout << gemm_kernel_launch_function.first << std::endl;
        std::pair<float, float> gemm_kernel_profile_result{
            profile_gemm<float>(m, n, k, lda, ldb, ldc, gemm_kernel_launch_function.second,
                                fp32_abs_tol, fp32_rel_tol, num_repeats, num_warmups)};
        std::cout << std::endl;
    }
    return 0;
}