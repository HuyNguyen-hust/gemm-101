#include <iostream>
#include <hip/hip_runtime.h>

#include "cuda_utils.hpp"

// cuda_utils
void check_cuda_error(hipError_t err, const char *const func, const char *const file, int const line)
{
    if (err != hipSuccess)
    {
        std::cerr << "Cuda runtime error at:" << file << ":" << line << std::endl;
        std::cerr << "Error Message:" << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void check_last_cuda_error(const char *const file, int const line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "Cuda runtime error at:" << file << ":" << line << std::endl;
        std::cerr << "Error Message:" << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

// gemm_utils